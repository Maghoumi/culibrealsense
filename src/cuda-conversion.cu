
#include <hip/hip_runtime.h>
#ifdef RS_USE_CUDA

#include "cuda-conversion.cuh"

__global__ void kernel_unpack_yuy2_rgb8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i >= superPixCount)
		return;

	int idx = i * 4;

	uint8_t y0 = src[idx];
	uint8_t u0 = src[idx + 1];
	uint8_t y1 = src[idx + 2];
	uint8_t v0 = src[idx + 3];

	int16_t c = y0 - 16;
	int16_t d = u0 - 128;
	int16_t e = v0 - 128;

	int32_t t;
#define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

	int odx = i * 6;

	dst[odx] = clamp((298 * c + 409 * e + 128) >> 8);
	dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	dst[odx + 2] = clamp((298 * c + 516 * d + 128) >> 8);

	c = y1 - 16;

	dst[odx + 3] = clamp((298 * c + 409 * e + 128) >> 8);
	dst[odx + 4] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
	dst[odx + 5] = clamp((298 * c + 516 * d + 128) >> 8);

#undef clamp(x)
}

void rsimpl::unpack_yuy2_rgb8_cuda(const uint8_t* src, uint8_t* dst, int n) 
{
	// How many super pixels do we have?
	int superPix = n / 2;
	uint8_t *devSrc = nullptr;
	uint8_t *devDst = nullptr;

	cudaError_t result = cudaMalloc(&devSrc, superPix * sizeof(uint8_t) * 4);
	assert(result == cudaSuccess);

	result = cudaMalloc(&devDst, n * sizeof(uint8_t) * 3);
	assert(result == cudaSuccess);

	result = cudaMemcpy(devSrc, src, superPix * sizeof(uint8_t) * 4, cudaMemcpyHostToDevice);
	assert(result == cudaSuccess);

	int numBlocks = superPix / RS_CUDA_THREADS_PER_BLOCK;

	// Call the kernel
	kernel_unpack_yuy2_rgb8_cuda<<<numBlocks, RS_CUDA_THREADS_PER_BLOCK >>>(devSrc, devDst, superPix);
	result = cudaGetLastError();
	assert(result == cudaSuccess);

	// Copy back
	result = cudaMemcpy(dst, devDst, n * sizeof(uint8_t) * 3, cudaMemcpyDeviceToHost);
	assert(result == cudaSuccess);

	cudaFree(devSrc);
	cudaFree(devDst);
}

#endif